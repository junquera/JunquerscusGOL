#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>
#include <conio.h>

#include "header.h"

//  declaraci�n de la kernel. Se le introducen por par�metro dos arrays unidimensionales. 
// Se sacan los �ndices mediante la divisi�n entera y el m�dulo. 
__global__ void compKernel(int* celdas, int* nuevo, int sizeX, int sizeY)
{
	int thx = threadIdx.x;
	int i = thx % sizeX;
	int j = thx / sizeX;

	int xm = (sizeX + ((i - 1) % sizeX)) % sizeX;
	int xM = (i + 1) % sizeX;

	int ym = (sizeY + ((j - 1) % sizeY)) % sizeY;
	int yM = (j + 1) % sizeY;


	int vecinos =	(celdas[xm + yM *sizeX] + celdas[i + yM *sizeX] + celdas[xM + yM *sizeX]) +
					(celdas[xm + j*sizeX] + celdas[xM + j*sizeX]) +
					(celdas[xm + ym * sizeX] + celdas[i + ym * sizeX] + celdas[xM + ym*sizeX]);

	if ((vecinos == 2 && celdas[i + j*sizeX]) || vecinos == 3){
		nuevo[i + j*sizeX] = 1;
	}
	else{
		nuevo[i + j * sizeX] = 0;
	}
}

int main(int argc, char *argv[])
{
	int i, manual = 0;
	struct_grid grid;

	//	Inicializamos los n�mero aleatorios
	srand((int)time(NULL));

	gridInit(10, 10, &grid);
	printGrid(grid);
	juega(&grid, 1);

	printGrid(grid);

	system("pause");

	hipDeviceReset();

	return 0;
}

void juega(struct_grid *t, int manual){
	while (compruebaCasillas(t)){
		if (manual){
			printf("\nPresiona intro para continuar o \"c\" para parar la ejecuci�n...\n");

			int option;
			do{
				option = getch();
				if (option == 67 || option == 99)
					return;
			} while (option != 13);

		}
		else{
			SLEEP(1000);
		}
		printGrid(*t);
	}
}
void gridInit(int x, int y, struct_grid *t){
	int i, j;

	(*t).sizeX = x;
	(*t).sizeY = y;
	(*t).celdas = (int**)malloc(sizeof(int) * (*t).sizeX);

	for (i = 0; i < (*t).sizeX; i++)
		(*t).celdas[i] = (int*)malloc((*t).sizeY * sizeof(int));

	(*t).nuevo = (int**)malloc(sizeof(int) * (*t).sizeX);
	for (i = 0; i < (*t).sizeX; i++)
		(*t).nuevo[i] = (int*)malloc((*t).sizeY * sizeof(int));

	for (i = 0; i < (*t).sizeX; i++){
		for (j = 0; j < (*t).sizeY; j++){
			(*t).celdas[i][j] = 0;
			(*t).nuevo[i][j] = 0;
		}
	}

	for (i = 0; i < (*t).sizeX; i++)
		for (j = 0; j < (*t).sizeY; j++)
			(*t).celdas[i][j] = rand() % 2;

}

void printGrid(struct_grid t){

	CLRSCR();
	printf("Game of life [CPU]\n\n");

	int i, j;

	for (i = 0; i < t.sizeX; i++){
		printf("[");
		for (j = 0; j < t.sizeY; j++){

			if (t.celdas[i][j])
				printf("%c", VIVA);
			else
				printf("%c", MUERTA);
		}
		printf("]\n");
	}
}

int* convierte(int** matriz, int dimX, int dimY){
	int* vector = (int*)malloc(dimX * dimY * sizeof(int));
	for (int i = 0; i < dimX; i++)
		for (int j = 0; j < dimY; j++)
			vector[i + j*dimX] = matriz[i][j];
	return vector;
}

int** convierte(int* vector, int dimX, int dimY){
	int i;
	int** matriz = (int**)malloc(sizeof(int) * dimX);

	for (i = 0; i < dimX; i++)
		matriz[i] = (int*)malloc(dimY * sizeof(int));

	for (int i = 0; i < dimX; i++)
		for (int j = 0; j < dimY; j++)
			matriz[i][j] = vector[i + j*dimX];

	return matriz;
}

int compruebaCasillas(struct_grid *t){

	int i, j, vivo = 0;

	int *dev_tablero = 0;
	int *dev_nuevo = 0;
	int size = (*t).sizeX * (*t).sizeY;

	// Allocate GPU buffers
	hipMalloc((void**)&dev_tablero, size*sizeof(int));
	hipMalloc((void**)&dev_nuevo, size*sizeof(int));

	hipMemcpy(dev_tablero, convierte((*t).celdas, (*t).sizeX, (*t).sizeY), size*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_nuevo, convierte((*t).celdas, (*t).sizeX, (*t).sizeY), size*sizeof(int), hipMemcpyHostToDevice);

	// Launch a kernel on the GPU with one thread for each element.
	compKernel <<<1 ,(*t).sizeX * (*t).sizeY >>>(dev_tablero, dev_nuevo, (*t).sizeX, (*t).sizeY);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	hipDeviceSynchronize();


	int *a = (int*)malloc((*t).sizeX * (*t).sizeY * sizeof(int));
	int *b = (int*)malloc((*t).sizeX * (*t).sizeY * sizeof(int)); 
	hipMemcpy(a, dev_tablero, (*t).sizeX*(*t).sizeY*sizeof(int *),hipMemcpyDeviceToHost);
	hipMemcpy(b, dev_nuevo, (*t).sizeX*(*t).sizeY*sizeof(int *), hipMemcpyDeviceToHost);
	(*t).celdas = convierte(a, (*t).sizeX, (*t).sizeY);
	(*t).nuevo = convierte(b, (*t).sizeX, (*t).sizeY);

	updateGrid(t);

	return 1;
}

void updateGrid(struct_grid *t){
	int i, j;

	for (i = 0; i < (*t).sizeX; i++){
		for (j = 0; j < (*t).sizeY; j++){
			(*t).celdas[i][j] = (*t).nuevo[i][j];
		}
	}
}