#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>
#include <conio.h>

#include "header.h"

//  declaraci�n de la kernel. Se le introducen por par�metro dos arrays unidimensionales. 
// Se sacan los �ndices mediante la divisi�n entera y el m�dulo. 
__global__ void compKernel(int* celdas, int* nuevo, int sizeX, int sizeY)
{
	int thx = threadIdx.x;
	int i = thx % sizeX;
	int j = thx / sizeX;

	//comprobar si los de arriba, abajo, izquierda y derecha se exceden del l�mite
	int xm = (sizeX + ((i - 1) % sizeX)) % sizeX;
	int xM = (i + 1) % sizeX;

	int ym = (sizeY + ((j - 1) % sizeY)) % sizeY;
	int yM = (j + 1) % sizeY;

	//sumar todos los vecinos
	int vecinos =	(celdas[xm + yM *sizeX] + celdas[i + yM *sizeX] + celdas[xM + yM *sizeX]) +
					(celdas[xm + j*sizeX] + celdas[xM + j*sizeX]) +
					(celdas[xm + ym * sizeX] + celdas[i + ym * sizeX] + celdas[xM + ym*sizeX]);

	//Comprobaci�n de la vida de la c�lula
	if ((vecinos == 2 && celdas[i + j*sizeX]) || vecinos == 3){
		nuevo[i + j*sizeX] = 1;
	}
	else{
		nuevo[i + j * sizeX] = 0;
	}
}
//---------------------------------------------------------------------------------------------------//
int main(int argc, char *argv[])
{
	int i, manual = 0;
	struct_grid grid;

	//	Inicializamos los n�mero aleatorios
	srand((int)time(NULL));

	//Comprobaci�n del n�mero de par�metros 
	if (argc > 1){

		int posSize = -1;
		for (i = 0; i < argc; i++){
			if (!strcmp(argv[i], "-a")){
				posSize = i;
				break;
			}
			if (!strcmp(argv[i], "-m")){
				manual = 1;
			}
		}

		if (posSize == -1)
			return -1;

		//inicializa el grid y lo pinta una vez
		gridInit(atoi(argv[posSize + 1]), atoi(argv[posSize + 2]), &grid);
		printGrid(grid);

	}

	juega(&grid, manual);

	system("pause");

	hipDeviceReset();

	return 0;
}
//---------------------------------------------------------------------------------------------------//
//m�tdo jugar. Juega mientras la el m�todo casillas devuelva 1. Mientras sea manual, estar� pidiendo 
//		precionar, si no, dormir� un segundo y seguir� ejecutando.
void juega(struct_grid *t, int manual){
	while (compruebaCasillas(t)){
		if (manual){
			printf("\nPresiona intro para continuar o \"c\" para parar la ejecuci�n...\n");

			int option;
			do{
				option = getch();
				if (option == 67 || option == 99)
					return;
			} while (option != 13);

		}
		else{
			SLEEP(1000);
		}
		printGrid(*t);
	}
}
//---------------------------------------------------------------------------------------------------//
//Hace un malloc de las estructuras del grid, dados unos tama�os. 
void gridInit(int x, int y, struct_grid *t){
	int i, j;

	(*t).sizeX = x;
	(*t).sizeY = y;
	(*t).celdas = (int**)malloc(sizeof(int) * (*t).sizeX);

	for (i = 0; i < (*t).sizeX; i++)
		(*t).celdas[i] = (int*)malloc((*t).sizeY * sizeof(int));

	(*t).nuevo = (int**)malloc(sizeof(int) * (*t).sizeX);
	for (i = 0; i < (*t).sizeX; i++)
		(*t).nuevo[i] = (int*)malloc((*t).sizeY * sizeof(int));

	for (i = 0; i < (*t).sizeX; i++){
		for (j = 0; j < (*t).sizeY; j++){
			(*t).celdas[i][j] = 0;
			(*t).nuevo[i][j] = 0;
		}
	}

	for (i = 0; i < (*t).sizeX; i++)
		for (j = 0; j < (*t).sizeY; j++)
			(*t).celdas[i][j] = rand() % 2;

}
//---------------------------------------------------------------------------------------------------//
//Pinta una matriz
void printGrid(struct_grid t){

	CLRSCR();
	printf("Game of life [GPU simple]\n\n");

	int i, j;

	for (i = 0; i < t.sizeX; i++){
		printf("[");
		for (j = 0; j < t.sizeY; j++){

			if (t.celdas[i][j])
				printf("%c", VIVA);
			else
				printf("%c", MUERTA);
		}
		printf("]\n");
	}
}
//---------------------------------------------------------------------------------------------------//
//Convierte una matriz bidimensional a una unidimensional
int* convierte(int** matriz, int dimX, int dimY){
	int* vector = (int*)malloc(dimX * dimY * sizeof(int));
	for (int i = 0; i < dimX; i++)
		for (int j = 0; j < dimY; j++)
			vector[i + j*dimX] = matriz[i][j];
	return vector;
}
//---------------------------------------------------------------------------------------------------//
//Convierte un array unidimensional a bidimensional
int** convierte(int* vector, int dimX, int dimY){
	int i;
	int** matriz = (int**)malloc(sizeof(int) * dimX);

	for (i = 0; i < dimX; i++)
		matriz[i] = (int*)malloc(dimY * sizeof(int));

	for (int i = 0; i < dimX; i++)
		for (int j = 0; j < dimY; j++)
			matriz[i][j] = vector[i + j*dimX];

	return matriz;
}
//---------------------------------------------------------------------------------------------------//
//Recibe una estructura struct_grid. Es la encargada de llamar al kernel. Reserva memoria para los arrays que enviar� al kernel
int compruebaCasillas(struct_grid *t){

	int i, j, vivo = 0;

	int *dev_tablero = 0;
	int *dev_nuevo = 0;
	int size = (*t).sizeX * (*t).sizeY;

	// reservar memoria CUDA para los tableros
	hipMalloc((void**)&dev_tablero, size*sizeof(int));
	hipMalloc((void**)&dev_nuevo, size*sizeof(int));

	hipMemcpy(dev_tablero, convierte((*t).celdas, (*t).sizeX, (*t).sizeY), size*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_nuevo, convierte((*t).celdas, (*t).sizeX, (*t).sizeY), size*sizeof(int), hipMemcpyHostToDevice);

	// Lanzar el kernel con un hilo por cada casilla
	compKernel <<<1 ,(*t).sizeX * (*t).sizeY >>>(dev_tablero, dev_nuevo, (*t).sizeX, (*t).sizeY);

	//Espera a que todos los hilos se sincronizen 
	hipDeviceSynchronize();

	//
	int *a = (int*)malloc((*t).sizeX * (*t).sizeY * sizeof(int));
	int *b = (int*)malloc((*t).sizeX * (*t).sizeY * sizeof(int)); 
	hipMemcpy(a, dev_tablero, (*t).sizeX*(*t).sizeY*sizeof(int *),hipMemcpyDeviceToHost);
	hipMemcpy(b, dev_nuevo, (*t).sizeX*(*t).sizeY*sizeof(int *), hipMemcpyDeviceToHost);
	(*t).celdas = convierte(a, (*t).sizeX, (*t).sizeY);
	(*t).nuevo = convierte(b, (*t).sizeX, (*t).sizeY);

	updateGrid(t);

	return 1;
}
//---------------------------------------------------------------------------------------------------//
void updateGrid(struct_grid *t){
	int i, j;

	for (i = 0; i < (*t).sizeX; i++){
		for (j = 0; j < (*t).sizeY; j++){
			(*t).celdas[i][j] = (*t).nuevo[i][j];
		}
	}
}
//---------------------------------------------------------------------------------------------------//